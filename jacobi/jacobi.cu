#include "hip/hip_runtime.h"
/**
 * jacobi2D.cu: This file is part of the PolyBench/GPU 1.0 test suite.
 *
 *
 * Contact: Scott Grauer-Gray <sgrauerg@gmail.com>
 * Will Killian <killian@udel.edu>
 * Louis-Noel Pouchet <pouchet@cse.ohio-state.edu>
 * Web address: http://www.cse.ohio-state.edu/~pouchet/software/polybench/GPU
 */

#include <stdio.h>
#include <unistd.h>
#include <time.h>
#include <sys/time.h>
#include <string.h>
#include <stdlib.h>
#include <stdarg.h>
#include <math.h>
#include "jacobi2D.cuh"

/* Problem size. */
#define TSTEPS 5
#define Ni 1000
#define Nj 1100
#define Nk 1200

#define DATATYPE = double
#define RUN_ON_CPU
using namespace std;


void init_array(array A, array B)
{
	int i, j;
	for (i = 0; i < n; i++)
	{
		for (j = 0; j < n; j++)
		{
			A[i][j] = ((DATA_TYPE) i*(j+2) / N);
			B[i][j] = ((DATA_TYPE) i*(j+3) / N);
		}
	}
    return A, B;
}


void runJacobi2DCpu(int tsteps, int n, DATA_TYPE POLYBENCH_2D(A,N,N,n,n), DATA_TYPE POLYBENCH_2D(B,N,N,n,n))
{
	for (int t = 0; t < TSTEPS; t++)
	{
    	for (int i = 1; i < N - 1; i++)
		{
			for (int j = 1; j < N - 1; j++)
			{
	  			B[i][j] = 0.2f * (A[i][j] + A[i][(j-1)] + A[i][(1+j)] + A[(1+i)][j] + A[(i-1)][j]);
			}
		}
		
    	for (int i = 1; i < N-1; i++)
		{
			for (int j = 1; j < N-1; j++)
			{
	  			A[i][j] = B[i][j];
			}
		}
	}
}


__global__ void runJacobiCUDA_kernel1(int n, DATA_TYPE* A, DATA_TYPE* B)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if ((i >= 1) && (i < (N-1)) && (j >= 1) && (j < (N-1)))
	{
		B[i*N + j] = 0.2f * (A[i*N + j] + A[i*N + (j-1)] + A[i*N + (1 + j)] + A[(1 + i)*N + j] + A[(i-1)*N + j]);	
	}
}

__global__ void runJacobiCUDA_kernel2(int n, DATA_TYPE* A, DATA_TYPE* B)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((i >= 1) && (i < (N-1)) && (j >= 1) && (j < (N-1)))
	{
		A[i*N + j] = B[i*N + j];
	}
}

void compareResults(int n, DATA_TYPE POLYBENCH_2D(a,N,N,n,n), DATA_TYPE POLYBENCH_2D(a_outputFromGpu,N,N,n,n), DATA_TYPE POLYBENCH_2D(b,N,N,n,n), DATA_TYPE POLYBENCH_2D(b_outputFromGpu,N,N,n,n))
{
	int i, j, fail;
	fail = 0;   

	// Compare output from CPU and GPU
	for (i=0; i<n; i++) 
	{
		for (j=0; j<n; j++) 
		{
			if (percentDiff(a[i][j], a_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
				fail++;
			}
        }
	}
  
	for (i=0; i<n; i++) 
	{
       	for (j=0; j<n; j++) 
		{
        		if (percentDiff(b[i][j], b_outputFromGpu[i][j]) > PERCENT_DIFF_ERROR_THRESHOLD) 
			{
        			fail++;
        		}
       	}
	}

	// Print results
	printf("Non-Matching CPU-GPU Outputs Beyond Error Threshold of %4.2f Percent: %d\n", PERCENT_DIFF_ERROR_THRESHOLD, fail);
}

void runCUDA(int tsteps, int n, DATA_TYPE A, DATA_TYPE B, DATA_TYPE A_outputFromGpu, DATA_TYPE B_outputFromGpu)
{
	DATA_TYPE* Agpu;
	DATA_TYPE* Bgpu;

	hipMalloc(&Agpu, N * N * sizeof(DATA_TYPE));
	hipMalloc(&Bgpu, N * N * sizeof(DATA_TYPE));
	hipMemcpy(Agpu, A, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);
	hipMemcpy(Bgpu, B, N * N * sizeof(DATA_TYPE), hipMemcpyHostToDevice);

	dim3 block(DIM_THREAD_BLOCK_X, DIM_THREAD_BLOCK_Y);
	dim3 grid((unsigned int)ceil( ((float)N) / ((float)block.x) ), (unsigned int)ceil( ((float)N) / ((float)block.y) ));
	
	/* Start timer. */

	for (int t = 0; t < TSTEPS; t++)
	{
		runJacobiCUDA_kernel1<<<grid,block>>>(n, Agpu, Bgpu);
		hipDeviceSynchronize();
		runJacobiCUDA_kernel2<<<grid,block>>>(n, Agpu, Bgpu);
		hipDeviceSynchronize();
	}

	/* Stop timer. */
    elapsed =
	
	hipMemcpy(A_outputFromGpu, Agpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);
	hipMemcpy(B_outputFromGpu, Bgpu, sizeof(DATA_TYPE) * N * N, hipMemcpyDeviceToHost);

	hipFree(Agpu);
	hipFree(Bgpu);

    return elepsed;
}

int main(int argc, char** argv)
{
	/* Retrieve problem size. */
	int n = N;
	int tsteps = TSTEPS;

    DATATYPE A[n][n];
    DATATYPE B[n][n];

	A, B = init_array(int n, array A, array B);

    //if GPU run = true
        //START TIMER
        runCUDA(tsteps, n, A, B, a_outputFromGpu, b_outputFromGpu);
        //end timer
    //if CPU run = true
        //start timer
        runCPU(tsteps, n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(b));
        //end timer
	
	compareResults(n, POLYBENCH_ARRAY(a), POLYBENCH_ARRAY(a_outputFromGpu), POLYBENCH_ARRAY(b), POLYBENCH_ARRAY(b_outputFromGpu));


	FREE_ARRAY(a);
	FREE_ARRAY(a_outputFromGpu);
	FREE_ARRAY(b);
	FREE_ARRAY(b_outputFromGpu);

	return 0;
}